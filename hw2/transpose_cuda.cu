#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include "transpose_cuda.cuh"

/**
 * TODO for all kernels (including naive):
 * Leave a comment above all non-coalesced memory accesses and bank conflicts.
 * Make it clear if the suboptimal access is a read or write. If an access is
 * non-coalesced, specify how many cache lines it touches, and if an access
 * causes bank conflicts, say if its a 2-way bank conflict, 4-way bank
 * conflict, etc.
 *
 * Comment all of your kernels.
*/


/**
 * Each block of the naive transpose handles a 64x64 block of the input matrix,
 * with each thread of the block handling a 1x4 section and each warp handling
 * a 32x4 section.
 *
 * If we split the 64x64 matrix into 32 blocks of shape (32, 4), then we have
 * a block matrix of shape (2 blocks, 16 blocks).
 * Warp 0 handles block (0, 0), warp 1 handles (1, 0), warp 2 handles (0, 1),
 * warp n handles (n % 2, n / 2).
 *
 * This kernel is launched with block shape (64, 16) and grid shape
 * (n / 64, n / 64) where n is the size of the square matrix.
 *
 * You may notice that we suggested in lecture that threads should be able to
 * handle an arbitrary number of elements and that this kernel handles exactly
 * 4 elements per thread. This is OK here because to overwhelm this kernel
 * it would take a 4194304 x 4194304  matrix, which would take ~17.6TB of
 * memory (well beyond what I expect GPUs to have in the next few years).
 */
__global__
void naiveTransposeKernel(const float *input, float *output, int n) {
  // TODO: do not modify code, just comment on suboptimal accesses

  const int i = threadIdx.x + 64 * blockIdx.x;
  int j = 4 * threadIdx.y + 64 * blockIdx.y;
  const int end_j = j + 4;

  /*
  Each warp handles a 32 x 4 submatrix and each thread in a warp
  handles a column of this submatrix. So a warp access 32 different
  columns. Thus the read is not coalesced since a warp will access
  32 different cache lines.
   */
  for (; j < end_j; j++) {
    output[j + n * i] = input[i + n * j];
  }
}

__global__
void shmemTransposeKernel(const float *input, float *output, int n) {
  // TODO: Modify transpose kernel to use shared memory. All global memory
  // reads and writes should be coalesced. Minimize the number of shared
  // memory bank conflicts (0 bank conflicts should be possible using
  // padding). Again, comment on all sub-optimal accesses.

  // Shared memory will store a 64x64 submatrix and be padded by a column at
  // the end since we will be accessing the shared memory stride 65 to avoid
  // memory bank conflicts
  __shared__ float data[65*64];

  const int i = threadIdx.x + 64 * blockIdx.x;
  int j = 4 * threadIdx.y + 64 * blockIdx.y;
  const int end_j = j + 4;

  const int i_t = threadIdx.x + 64 * blockIdx.y;
  int j_t = 4 * threadIdx.y + 64 * blockIdx.x;
  const int end_j_t = j_t + 4;

  const int i_data = threadIdx.x;
  int j_data = 4 * threadIdx.y;
  for (; j < end_j; j++) {
    data[j_data + 65*i_data] = input[i + n * j];
    j_data++;
  }
  __syncthreads();
  j_data -= 4;
  for (; j_t < end_j_t; j_t++) {
    output[i_t + n * j_t] = data[i_data + 65 * j_data];
    j_data++;
  }

}

__global__
void optimalTransposeKernel(const float *input, float *output, int n) {
  // TODO: This should be based off of your shmemTransposeKernel.
  // Use any optimization tricks discussed so far to improve performance.
  // Consider ILP and loop unrolling.

  const int i = threadIdx.x + 64 * blockIdx.x;
  int j = 4 * threadIdx.y + 64 * blockIdx.y;
  const int end_j = j + 4;

  for (; j < end_j; j++) {
    output[j + n * i] = input[i + n * j];
  }
}

void cudaTranspose(const float *d_input,
                   float *d_output,
                   int n,
                   TransposeImplementation type) {
  if (type == NAIVE) {
    dim3 blockSize(64, 16);
    dim3 gridSize(n / 64, n / 64);
    naiveTransposeKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
  } else if (type == SHMEM) {
    dim3 blockSize(64, 16);
    dim3 gridSize(n / 64, n / 64);
    shmemTransposeKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
  } else if (type == OPTIMAL) {
    dim3 blockSize(64, 16);
    dim3 gridSize(n / 64, n / 64);
    optimalTransposeKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
  } else {
    // unknown type
    assert(false);
  }
}
