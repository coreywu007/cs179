#include "hip/hip_runtime.h"

/* 
Based off work by Nelson, et al.
Brigham Young University (2010)

Adapted by Kevin Yuh (2015)
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <cmath>
#define PI 3.14159265358979

/* Check errors on CUDA runtime functions */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

texture<float,2,hipReadModeElementType> texreference;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

/* Check errors on cuFFT functions */
void gpuFFTchk(int errval){
    if (errval != HIPFFT_SUCCESS){
        printf("Failed FFT call, error code %d\n", errval);
    }
}

/* Check errors on CUDA kernel calls */
void checkCUDAKernelError()
{
    hipError_t err = hipGetLastError();
    if  (hipSuccess != err){
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    } else {
        fprintf(stderr, "No kernel error detected\n");
    }

}

/* Extract real part of complex numbers */
__global__ void cudaExtractReal(hipfftComplex *dev_sinogram_cmplx,
        float *dev_sinogram, const int totalSize) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    while(i < totalSize) {
        dev_sinogram[i] = dev_sinogram_cmplx[i].x;
        i += blockDim.x * gridDim.x;
    }
 
}

/* Perform ramp filter for complex signal */
__global__ void cudaFrequencyScaleKernel(hipfftComplex *dev_sinogram_cmplx,
    const int sinogram_width, const int totalSize) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float sinogram_center =  ((sinogram_width - 1)/ 2.0);
    // For a given angle, scaling factor is 1 - dist_from_center / (n/2)
    while(i < totalSize) {
        // Calculate dist from center
        int dist_from_center = abs((float)(i % sinogram_width - sinogram_center));
        // Calculate scaling factor
        float scalingFactor = (1.0 - (float) dist_from_center / sinogram_center); 
        dev_sinogram_cmplx[i].x *= scalingFactor;
        dev_sinogram_cmplx[i].y *= scalingFactor;
        i += blockDim.x * gridDim.x;
    }
}

/* Perform back projection to reconstruct image from sinogram data */
__global__ void cudaBackProjection(float *output_dev, float *dev_sinogram, const int sinogram_width,
    const int nAngles, const int width, const int height,const int mid_width,
    const int mid_height,const float mid_sinogram) {

    int x = blockIdx.x * blockDim.x + threadIdx.x; // pixel coord
    int y = blockIdx.y * blockDim.y + threadIdx.y; // pixel coord
    float x_geo, y_geo; // geometric coord
    float x_i, y_i; // intersection point
    float theta, m, q, d;
    // Don't go past bounds of image. Paralellize over pixels
    for(;x < width; x += blockDim.x * gridDim.x) {
        for(; y < height; y += blockDim.y * gridDim.y) {
            for(int thetaNo = 0 ; thetaNo < nAngles; thetaNo++) {
                // Calculate theta based on angle number
                theta = thetaNo * PI / nAngles;
                // Convert pixel coord to geometric coordinate
                x_geo = x - mid_width;
                y_geo = mid_height - y;
                // Handle edges cases where theta is at 0 or pi/2
                if (theta == 0) {
                    d = x_geo;
                }
                else if (theta == PI/2) {
                    d = y_geo;
                }
                else {
                     // Calculate slope from theta
                    m = -1.0f/tan(theta);
                    q = -1.0f/m;
                    // Find intersection point
                    x_i = (y_geo - m*x_geo) / (q - m);
                    y_i = q*x_i;
                    // Calculate distance
                    d = sqrtf((x_i*x_i + y_i*y_i));
                    // Use -d instead of d when q > 0 and x_i < 0 or if -1/m < 0 and x_i > 0
                    if ((q > 0 && x_i < 0) || (q < 0 && x_i > 0))
                        d = -d;               
                }
                // Use texture memory to read from sinogram data
                output_dev[y*width + x] += tex2D(texreference, (int)mid_sinogram + (int)d, thetaNo);
                // To use global memory instead of texture memory, comment the previous
                // line and uncomment the next line.
                //output_dev[y*width + x] += dev_sinogram[(int)mid_sinogram +(int) d + thetaNo *sinogram_width];
            }
        }
    }
}


int main(int argc, char** argv){
    hipSetDevice(1);
    printf("Starting program\n");
    if (argc != 7){
        fprintf(stderr, "Incorrect number of arguments.\n\n");
        fprintf(stderr, "\nArguments: \n \
        < Sinogram filename > \n \
        < Width or height of original image, whichever is larger > \n \
        < Number of angles in sinogram >\n \
        < threads per block >\n \
        < number of blocks >\n \
        < output filename >\n");
        exit(EXIT_FAILURE);
    }

    /********** Parameters **********/

    int width = atoi(argv[2]);
    int height = width;
    int sinogram_width = (int)ceilf( height * sqrt(2) );
    int nAngles = atoi(argv[3]);

    int threadsPerBlock = atoi(argv[4]);
    int nBlocks = atoi(argv[5]);

    int sinogram_cmplx_byte_size = (sinogram_width*nAngles*sizeof(hipfftComplex));
    int sinogram_byte_size = sinogram_width*nAngles*sizeof(float);

    int mid_width = (int) floor(width / 2.0);
    int mid_height = (int) floor(height / 2.0);
    // Get true mid point by subtracting 1 since we're zero indexing
    int mid_sinogram_width = (int) floor((sinogram_width-1)/2.0);
    /********** Data storage *********/

    // GPU DATA STORAGE
    hipfftComplex *dev_sinogram_cmplx;
    float* dev_sinogram_float; 
    float* output_dev;  // Image storage

    // Texture data storage
    dim3 blocknum;
    dim3 blocksize;
    hipArray* carray;
    hipChannelFormatDesc channel;

    // Host data storage
    hipfftComplex *sinogram_host;
    size_t size_result = width*height*sizeof(float);
    float *output_host = (float *)malloc(size_result);

    /*********** Set up IO, Read in data ************/
    sinogram_host = (hipfftComplex *)malloc(  sinogram_width *nAngles * sizeof(hipfftComplex) );

    FILE *dataFile = fopen(argv[1],"r");
    if (dataFile == NULL){
        fprintf(stderr, "Sinogram file missing\n");
        exit(EXIT_FAILURE);
    }

    FILE *outputFile = fopen(argv[6], "w");
    if (outputFile == NULL){
        fprintf(stderr, "Output file cannot be written\n");
        exit(EXIT_FAILURE);
    }

    int j, i;
    for(i = 0; i < nAngles * sinogram_width; i++){
        fscanf(dataFile,"%f",&sinogram_host[i].x);
        sinogram_host[i].y = 0;
    }

    fclose(dataFile);

    /*********** Assignment starts here *********/

    /* TODO: Allocate memory for all GPU storage above, copy input sinogram
    over to dev_sinogram_cmplx. */
    hipMalloc((void **)&dev_sinogram_cmplx, sinogram_cmplx_byte_size);
    hipMalloc((void **)&dev_sinogram_float, sinogram_byte_size);

    gpuErrchk( hipMemcpy(dev_sinogram_cmplx, sinogram_host, sinogram_cmplx_byte_size,
                 hipMemcpyHostToDevice));

    /* Implement the high-pass filter:
        - Use cuFFT for the forward FFT
        - Create your own kernel for the frequency scaling.
        - Use cuFFT for the inverse FFT
        - extract real components to floats
        - Free the original sinogram (dev_sinogram_cmplx)

        Note: If you want to deal with real-to-complex and complex-to-real
        transforms in cuFFT, you'll have to slightly change our code above.
    */
   
    /* Create a cuFFT plan for the forward transform. */
    hipfftHandle plan;
    int batch = nAngles; // Number of transforms to run
    hipfftPlan1d(&plan, sinogram_width, HIPFFT_C2C, batch);

    /* Run the forward DFT on the input signal in-place */
    hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_FORWARD);

    /* Call frequency scaling kernel */
    cudaFrequencyScaleKernel<<<nBlocks, threadsPerBlock>>>(dev_sinogram_cmplx, 
        sinogram_width, sinogram_width*nAngles);

    /* Run backward DFT on output signal and extract real part */
    hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_BACKWARD);

    cudaExtractReal<<<nBlocks, threadsPerBlock>>>( dev_sinogram_cmplx, 
        dev_sinogram_float, sinogram_width*nAngles);

    // Free dev data and destroy plan
    hipFree(dev_sinogram_cmplx);
    hipfftDestroy(plan);

    /* Implement backprojection.
        - Allocate memory for the output image.
        - Create your own kernel to accelerate backprojection.
        - Copy the reconstructed image back to output_host.
        - Free all remaining memory on the GPU.
    */
    hipMalloc((void **)&output_dev, size_result);
    hipMemset(output_dev, 0, size_result);

    /* Set up texture memory */
    channel = hipCreateChannelDesc<float>();
    hipMallocArray(&carray, &channel, sinogram_width, nAngles);
    hipMemcpyToArray(carray, 0, 0, dev_sinogram_float, sinogram_byte_size, hipMemcpyDeviceToDevice);

    // Set texture filter mode property to linear and address mode to clamp
    texreference.normalized = 0;
    texreference.filterMode = hipFilterModeLinear;
    texreference.addressMode[0] = hipAddressModeClamp;
    texreference.addressMode[1] = hipAddressModeClamp;

    // Bind texture to array
    hipBindTextureToArray(texreference,carray);

    // Block size will be 16x16
    // Use necessary grid size to cover image
    blocksize.x=16;
    blocksize.y=16;
    blocknum.x=(int) ceil((float)width/16);
    blocknum.y=(int) ceil((float)height/16);

    printf("Starting back projection\n");
    cudaBackProjection<<<blocknum, blocksize>>>(output_dev, 
        dev_sinogram_float, sinogram_width, nAngles, width, height,
        mid_width, mid_height, mid_sinogram_width);

    //Unbind texture reference to free resource
    hipUnbindTexture(texreference);

    // Copy result matrix from device to host
    hipMemcpy( output_host, output_dev, size_result, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_sinogram_float);
    hipFree(output_dev);
    hipFreeArray(carray);
    /* Export image data. */

    for(j = 0; j < width; j++){
        for(i = 0; i < height; i++){
            fprintf(outputFile, "%e ",output_host[j*width + i]);
        }
        fprintf(outputFile, "\n");
    }


    /* Cleanup: Free host memory, close files. */
    free(sinogram_host);
    free(output_host);

    fclose(outputFile);

    return 0;
}


