#include "hip/hip_runtime.h"

/* 
Based off work by Nelson, et al.
Brigham Young University (2010)

Adapted by Kevin Yuh (2015)
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#define PI 3.14159265358979

/* Check errors on CUDA runtime functions */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

texture<float,2,hipReadModeElementType> texreference;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

/* Check errors on cuFFT functions */
void gpuFFTchk(int errval){
    if (errval != HIPFFT_SUCCESS){
        printf("Failed FFT call, error code %d\n", errval);
    }
}

/* Check errors on CUDA kernel calls */
void checkCUDAKernelError()
{
    hipError_t err = hipGetLastError();
    if  (hipSuccess != err){
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    } else {
        fprintf(stderr, "No kernel error detected\n");
    }

}

__global__ void cudaExtractReal(hipfftComplex *dev_sinogram_cmplx, float *dev_sinogram, const int totalSize) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    while(i < totalSize) {
        dev_sinogram[i] = dev_sinogram_cmplx[i].x;
        i += blockDim.x * gridDim.x;
    }
 
}

void cudaCallExtractReal(const unsigned int nBlocks, const unsigned int threadsPerBlock,
        hipfftComplex *dev_sinogram_cmplx, float *dev_sinogram_float, const int totalSize){

    cudaExtractReal<<<nBlocks, threadsPerBlock>>>( dev_sinogram_cmplx, dev_sinogram_float, totalSize);
}

__global__ void cudaFrequencyScaleKernel(hipfftComplex *dev_sinogram_cmplx,
    const int sinogram_width, const int nAngles) {

    const int totalSize = nAngles * sinogram_width;

    /*Divide all data by the value pointed to by max_abs_val. */
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // For a given angle, scaling factor is 1 - dist_from_center / (n/2)
    // = 1 - abs(n/2 - i) / (n/2) = 1 - abs(1 - 2*i/n)
    float scalingFactor = 1 - fabsf(1 - 2*(i % sinogram_width)) / sinogram_width;
    //float scalingFactor = 0.5; 
    while(i < totalSize) {
        dev_sinogram_cmplx[i].x *= scalingFactor;
        dev_sinogram_cmplx[i].y *= scalingFactor;
        i += blockDim.x * gridDim.x;
    }
}


void cudaCallFrequencyScaleKernel(const unsigned int blocks, const unsigned int threadsPerBlock,
    hipfftComplex *dev_sinogram_cmplx, const int sinogram_width, const int nAngles) {

    //cudaFrequencyScaleKernel<<<blocks, threadsPerBlock>>>(dev_sinogram_cmplx, sinogram_width, nAngles);

}

__global__ void cudaBackProjection(float *output_dev, float *dev_sinogram, const int sinogram_width,
    const int nAngles, const int width, const int height, const float theta_step,const int mid_width,
    const int mid_height,const int mid_sinogram_width) {

    int x = blockIdx.x * blockDim.x + threadIdx.x; // pixel coord
    int y = blockIdx.y * blockDim.y + threadIdx.y; // pixel coord
    int x_geo = x - mid_width;
    int y_geo = y - mid_height;
    float x_i, y_i;
    int d;
    float theta, m, q;
    //for(;x < width; x += blockDim.x * gridDim.x) {
    //    for(; y < height; y += blockDim.y * gridDim.y) {
    for(int thetaNo = 0 ; thetaNo < nAngles; thetaNo++) {
        // Calculate theta based on angle number
        theta = (float)thetaNo * theta_step;

        if (theta == 0) {
            d = x_geo;
        }
        else if (theta == PI/2) {
            d = y_geo;
        }
        else if (theta == PI) {
            d = -x_geo;
        }
        else if (theta == 3*PI/2) {
            d = -y_geo;
        }
        else {
             // Calculate slope from theta
            m = -cos(theta)/sin(theta);
            q = -1.0/m;
            // Handle edge cases
            x_i = (float)(y_geo - m*x_geo) / (q - m);
            y_i = q*x_i;
            d = (int) sqrtf((x_i*x_i + y_i*y_i));

            // Use -d instead of d when x_i < 0 or if -1/m < 0 and x_i ? 0
            if (x_i < 0 || (q < 0 && x_i > 0))
                d = -d;               
        }
        //output_dev[y*width + x] += tex2D(texreference, mid_sinogram_width + d, thetaNo);
        output_dev[y*width + x] += dev_sinogram[mid_sinogram_width + d + thetaNo *sinogram_width];
    }
    //    }
    //}
}

void cudaCallBackProjection(const dim3 blocknum, const dim3 blocksize, 
    float *output_dev, float *dev_sinogram_float, 
    const int sinogram_width, const int nAngles,const int width,const int height,
    const float theta_step, const int mid_width,const int mid_height,const int mid_sinogram_width) {
    cudaBackProjection<<<blocknum, blocksize>>>(output_dev, 
        dev_sinogram_float, sinogram_width, nAngles, width, height, theta_step,
        mid_width, mid_height, mid_sinogram_width);
}



int main(int argc, char** argv){
    printf("Starting program\n");
    if (argc != 7){
        fprintf(stderr, "Incorrect number of arguments.\n\n");
        fprintf(stderr, "\nArguments: \n \
        < Sinogram filename > \n \
        < Width or height of original image, whichever is larger > \n \
        < Number of angles in sinogram >\n \
        < threads per block >\n \
        < number of blocks >\n \
        < output filename >\n");
        exit(EXIT_FAILURE);
    }

    /********** Parameters **********/

    int width = atoi(argv[2]);
    int height = width;
    int sinogram_width = (int)ceilf( height * sqrt(2) );
    int nAngles = atoi(argv[3]);

    int threadsPerBlock = atoi(argv[4]);
    int nBlocks = atoi(argv[5]);

    int sinogram_cmplx_byte_size = (sinogram_width*nAngles*sizeof(hipfftComplex));
    int sinogram_byte_size = sinogram_width*nAngles*sizeof(float);
    float theta_step = PI / (float) nAngles;
    int mid_width = (int) floor((float) width / 2.0);
    int mid_height = (int) floor((float) height / 2.0);
    int mid_sinogram_width = (int) floor((float) sinogram_width/2.0);
    /********** Data storage *********/

    // GPU DATA STORAGE
    hipfftComplex *dev_sinogram_cmplx;
    float *dev_sinogram_float; 
    float* output_dev;  // Image storage

    // Texture data storage
    dim3 blocknum;
    dim3 blocksize;
    hipArray* carray;
    hipChannelFormatDesc channel;

    // Host data storage
    hipfftComplex *sinogram_host;
    size_t size_result = width*height*sizeof(float);
    float *output_host = (float *)malloc(size_result);
    float *sinogram_float = (float *)malloc(sinogram_byte_size);

    /*********** Set up IO, Read in data ************/
    printf("sinogram_width: %d, theta_step: %f, mid_width, mid_height, mid_sinogram_width: %d %d %d\n",
            sinogram_width, theta_step, mid_width, mid_height, mid_sinogram_width);
    sinogram_host = (hipfftComplex *)malloc(  sinogram_width *nAngles * sizeof(hipfftComplex) );

    FILE *dataFile = fopen(argv[1],"r");
    if (dataFile == NULL){
        fprintf(stderr, "Sinogram file missing\n");
        exit(EXIT_FAILURE);
    }

    FILE *outputFile = fopen(argv[6], "w");
    if (outputFile == NULL){
        fprintf(stderr, "Output file cannot be written\n");
        exit(EXIT_FAILURE);
    }

    int j, i;
    printf("Reading input sinogram file\n");
    for(i = 0; i < nAngles * sinogram_width; i++){
        fscanf(dataFile,"%f",&sinogram_host[i].x);
        sinogram_host[i].y = 0;
    }

    fclose(dataFile);
    printf("Finished reading input sinogram file\n");

    /*********** Assignment starts here *********/

    /* TODO: Allocate memory for all GPU storage above, copy input sinogram
    over to dev_sinogram_cmplx. */
    hipMalloc((void **)&dev_sinogram_cmplx, sinogram_cmplx_byte_size);
    hipMalloc((void **)&dev_sinogram_float, sinogram_byte_size);

    printf("Copying sinogram from host to device for high pass filter\n");
    gpuErrchk( hipMemcpy(dev_sinogram_cmplx, sinogram_host, sinogram_cmplx_byte_size,
                 hipMemcpyHostToDevice));

    /* TODO 1: Implement the high-pass filter:
        - Use cuFFT for the forward FFT
        - Create your own kernel for the frequency scaling.
        - Use cuFFT for the inverse FFT
        - extract real components to floats
        - Free the original sinogram (dev_sinogram_cmplx)

        Note: If you want to deal with real-to-complex and complex-to-real
        transforms in cuFFT, you'll have to slightly change our code above.
    */
   
    /* Create a cuFFT plan for the forward transform. */
    hipfftHandle plan;
    int batch = nAngles; // Number of transforms to run

    hipfftPlan1d(&plan, sinogram_cmplx_byte_size, HIPFFT_C2C, batch);
    /* Run the forward DFT on the input signal in-place */
    hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_FORWARD);

    printf("Executing frequency scale kernel\n");
    /* Call frequency scaling kernel */
    cudaCallFrequencyScaleKernel(nBlocks, threadsPerBlock, dev_sinogram_cmplx, sinogram_width, nAngles);
    printf("Finished executing scale kernel\n");

    /* Create new cuFFT plan for backward transform. */
    //hipfftPlan1d(&plan, sinogram_cmplx_byte_size, HIPFFT_C2R, batch);
    /* Run backward DFT on output signal and extract real part */
    //hipfftExecC2R(plan, dev_sinogram_cmplx, dev_sinogram_float);
    hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_BACKWARD);

    printf("Copying sinogram data data back to host\n");
    cudaCallExtractReal(nBlocks, threadsPerBlock, dev_sinogram_cmplx, dev_sinogram_float, sinogram_width*nAngles);
    /* Copy data back to host */
    hipMemcpy( sinogram_float, dev_sinogram_float, sinogram_byte_size, hipMemcpyDeviceToHost);
    hipFree(dev_sinogram_cmplx);


    /* TODO 2: Implement backprojection.
        - Allocate memory for the output image.
        - Create your own kernel to accelerate backprojection.
        - Copy the reconstructed image back to output_host.
        - Free all remaining memory on the GPU.
    */
    hipMalloc((void **)&output_dev, size_result);
    hipMemset(output_dev, 0, size_result);
    /* Set up texture memory */
    // Create channel to descibe data type
    channel = hipCreateChannelDesc<float>();
    hipMallocArray(&carray, &channel, sinogram_width, nAngles);
    // Copy sinogram from host to device
    hipMemcpyToArray(carray, 0, 0, sinogram_float, sinogram_byte_size, hipMemcpyHostToDevice);

    // Set texture filterm mode property to linear
    texreference.normalized = 0;
    texreference.filterMode = hipFilterModeLinear;
    // Set texture address mode to clamp
    texreference.addressMode[0] = hipAddressModeClamp;
    texreference.addressMode[1] = hipAddressModeClamp;
    // Bind texture reference with cuda array
    hipBindTextureToArray(texreference,carray);
    blocksize.x=16;
    blocksize.y=16;
    blocknum.x=(int) ceil((float)width/16);
    blocknum.y=(int) ceil((float)height/16);

    printf("Executing back projection kernel\n");
    cudaCallBackProjection(blocknum, blocksize, output_dev, 
        dev_sinogram_float, sinogram_width, nAngles, width, height,theta_step,
        mid_width, mid_height, mid_sinogram_width);
    printf("Finished executing back projection kernel\n");
    //Unbind texture reference to free resource
    hipUnbindTexture(texreference);

    // Copy result matrix from device to host
    hipMemcpy( output_host, output_dev, size_result, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_sinogram_float);
    hipFree(output_dev);
    hipFreeArray(carray);
    /* Export image data. */

    for(j = 0; j < width; j++){
        for(i = 0; i < height; i++){
            fprintf(outputFile, "%e ",output_host[j*width + i]);
        }
        fprintf(outputFile, "\n");
    }


    /* Cleanup: Free host memory, close files. */
    free(sinogram_float);
    free(sinogram_host);
    free(output_host);

    fclose(outputFile);

    return 0;
}


