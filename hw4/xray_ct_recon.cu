#include "hip/hip_runtime.h"

/* 
Based off work by Nelson, et al.
Brigham Young University (2010)

Adapted by Kevin Yuh (2015)
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <cmath>
#define PI 3.14159265358979

/* Check errors on CUDA runtime functions */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

texture<float,2,hipReadModeElementType> texreference;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

/* Check errors on cuFFT functions */
void gpuFFTchk(int errval){
    if (errval != HIPFFT_SUCCESS){
        printf("Failed FFT call, error code %d\n", errval);
    }
}

/* Check errors on CUDA kernel calls */
void checkCUDAKernelError()
{
    hipError_t err = hipGetLastError();
    if  (hipSuccess != err){
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    } else {
        fprintf(stderr, "No kernel error detected\n");
    }

}

__global__ void cudaExtractReal(hipfftComplex *dev_sinogram_cmplx,
        float *dev_sinogram, const int totalSize) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    while(i < totalSize) {
        dev_sinogram[i] = dev_sinogram_cmplx[i].x;
        i += blockDim.x * gridDim.x;
    }
 
}

__global__ void cudaFrequencyScaleKernel(hipfftComplex *dev_sinogram_cmplx,
    const int sinogram_width, const int totalSize) {

    /*Divide all data by the value pointed to by max_abs_val. */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float sinogram_center =  ((sinogram_width - 1)/ 2.0);
    // For a given angle, scaling factor is 1 - dist_from_center / (n/2)
    // = 1 - abs(n/2 - i) / (n/2) = 1 - abs(1 - 2*i/n)
    //float scalingFactor = 1.0 - fabsf(1.0 - 2.0*(i % sinogram_width) / sinogram_width);
    while(i < totalSize) {
        int dist_from_center = abs((float)(i % sinogram_width - sinogram_center));
        float scalingFactor = (1.0 - (float) dist_from_center / sinogram_center); 
        //printf("%d %f %d\n", i % sinogram_width, scalingFactor, dist_from_center);
        dev_sinogram_cmplx[i].x *= scalingFactor;
        dev_sinogram_cmplx[i].y *= scalingFactor;
        i += blockDim.x * gridDim.x;
    }
}

__global__ void cudaBackProjection(float *output_dev, float *dev_sinogram, const int sinogram_width,
    const int nAngles, const int width, const int height,const int mid_width,
    const int mid_height,const int mid_sinogram_width) {

    int x = blockIdx.x * blockDim.x + threadIdx.x; // pixel coord
    int y = blockIdx.y * blockDim.y + threadIdx.y; // pixel coord
    float x_geo, y_geo;
    float x_i, y_i;
    float theta, m, q, d;
    float mid = (sinogram_width - 1) / 2.0;
    for(;x < width; x += blockDim.x * gridDim.x) {
        for(; y < height; y += blockDim.y * gridDim.y) {
            for(int thetaNo = 0 ; thetaNo < nAngles; thetaNo++) {
                // Calculate theta based on angle number
                theta = thetaNo * PI / nAngles;
                x_geo = x - mid_width;
                y_geo = mid_height - y;
                if (theta <= 0.01) {
                    d = x_geo;
                }
                else if (abs(theta - PI/2) <= 0.01) {
                    d = y_geo;
                }
                else {
                     // Calculate slope from theta
                    m = -1.0f/tan(theta);
                    q = -1.0f/m;
                    // Handle edge cases
                    x_i = (y_geo - m*x_geo) / (q - m);
                    y_i = q*x_i;
                    d = sqrtf((x_i*x_i + y_i*y_i));

                    // Use -d instead of d when x_i < 0 or if -1/m < 0 and x_i ? 0
                    if ((q > 0 && x_i < 0) || (q < 0 && x_i > 0))
                        d = -d;               
                }
                output_dev[y*width + x] += tex2D(texreference, mid + d, thetaNo);
                //output_dev[y*width + x] += dev_sinogram[(int)mid +(int) d + thetaNo *sinogram_width];
            }
        }
    }
}


int main(int argc, char** argv){
    hipSetDevice(1);
    printf("Starting program\n");
    if (argc != 7){
        fprintf(stderr, "Incorrect number of arguments.\n\n");
        fprintf(stderr, "\nArguments: \n \
        < Sinogram filename > \n \
        < Width or height of original image, whichever is larger > \n \
        < Number of angles in sinogram >\n \
        < threads per block >\n \
        < number of blocks >\n \
        < output filename >\n");
        exit(EXIT_FAILURE);
    }

    /********** Parameters **********/

    int width = atoi(argv[2]);
    int height = width;
    int sinogram_width = (int)ceilf( height * sqrt(2) );
    int nAngles = atoi(argv[3]);

    int threadsPerBlock = atoi(argv[4]);
    int nBlocks = atoi(argv[5]);

    int sinogram_cmplx_byte_size = (sinogram_width*nAngles*sizeof(hipfftComplex));
    int sinogram_byte_size = sinogram_width*nAngles*sizeof(float);

    int mid_width = (int) floor(width / 2.0);
    int mid_height = (int) floor(height / 2.0);
    int mid_sinogram_width = (int) floor(sinogram_width/2.0);
    /********** Data storage *********/

    // GPU DATA STORAGE
    hipfftComplex *dev_sinogram_cmplx;
    float* dev_sinogram_float; 
    float* output_dev;  // Image storage

    // Texture data storage
    dim3 blocknum;
    dim3 blocksize;
    hipArray* carray;
    hipChannelFormatDesc channel;

    // Host data storage
    hipfftComplex *sinogram_host;
    size_t size_result = width*height*sizeof(float);
    float *output_host = (float *)malloc(size_result);

    /*********** Set up IO, Read in data ************/
    sinogram_host = (hipfftComplex *)malloc(  sinogram_width *nAngles * sizeof(hipfftComplex) );

    FILE *dataFile = fopen(argv[1],"r");
    if (dataFile == NULL){
        fprintf(stderr, "Sinogram file missing\n");
        exit(EXIT_FAILURE);
    }

    FILE *outputFile = fopen(argv[6], "w");
    if (outputFile == NULL){
        fprintf(stderr, "Output file cannot be written\n");
        exit(EXIT_FAILURE);
    }

    int j, i;
    for(i = 0; i < nAngles * sinogram_width; i++){
        fscanf(dataFile,"%f",&sinogram_host[i].x);
        sinogram_host[i].y = 0;
    }

    fclose(dataFile);

    /*********** Assignment starts here *********/

    /* TODO: Allocate memory for all GPU storage above, copy input sinogram
    over to dev_sinogram_cmplx. */
    hipMalloc((void **)&dev_sinogram_cmplx, sinogram_cmplx_byte_size);
    hipMalloc((void **)&dev_sinogram_float, sinogram_byte_size);

    gpuErrchk( hipMemcpy(dev_sinogram_cmplx, sinogram_host, sinogram_cmplx_byte_size,
                 hipMemcpyHostToDevice));

    /* TODO 1: Implement the high-pass filter:
        - Use cuFFT for the forward FFT
        - Create your own kernel for the frequency scaling.
        - Use cuFFT for the inverse FFT
        - extract real components to floats
        - Free the original sinogram (dev_sinogram_cmplx)

        Note: If you want to deal with real-to-complex and complex-to-real
        transforms in cuFFT, you'll have to slightly change our code above.
    */
   
    /* Create a cuFFT plan for the forward transform. */
    hipfftHandle plan;
    int batch = nAngles; // Number of transforms to run
    hipfftPlan1d(&plan, sinogram_width, HIPFFT_C2C, batch);

    /* Run the forward DFT on the input signal in-place */
    hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_FORWARD);

    /* Call frequency scaling kernel */
    cudaFrequencyScaleKernel<<<nBlocks, threadsPerBlock>>>(dev_sinogram_cmplx, 
        sinogram_width, sinogram_width*nAngles);

    /* Run backward DFT on output signal and extract real part */
    hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_BACKWARD);

    cudaExtractReal<<<nBlocks, threadsPerBlock>>>( dev_sinogram_cmplx, 
        dev_sinogram_float, sinogram_width*nAngles);

    /* Copy data back to host */
    hipMemcpy( sinogram_host, dev_sinogram_float, sinogram_byte_size, hipMemcpyDeviceToHost);
    hipFree(dev_sinogram_cmplx);
    hipfftDestroy(plan);

    /* TODO 2: Implement backprojection.
        - Allocate memory for the output image.
        - Create your own kernel to accelerate backprojection.
        - Copy the reconstructed image back to output_host.
        - Free all remaining memory on the GPU.
    */
    hipMalloc((void **)&output_dev, size_result);
    hipMemset(output_dev, 0, size_result);

    /* Set up texture memory */
    channel = hipCreateChannelDesc<float>();
    hipMallocArray(&carray, &channel, sinogram_width, nAngles);
    hipMemcpyToArray(carray, 0, 0, dev_sinogram_float, sinogram_byte_size, hipMemcpyDeviceToDevice);

    // Set texture filter mode property to linear and address mode to clamp
    texreference.normalized = 0;
    texreference.filterMode = hipFilterModeLinear;
    texreference.addressMode[0] = hipAddressModeClamp;
    texreference.addressMode[1] = hipAddressModeClamp;

    hipBindTextureToArray(texreference,carray);

    blocksize.x=16;
    blocksize.y=16;
    blocknum.x=(int) ceil((float)width/16);
    blocknum.y=(int) ceil((float)height/16);

    printf("Starting back projection\n");
    cudaBackProjection<<<blocknum, blocksize>>>(output_dev, 
        dev_sinogram_float, sinogram_width, nAngles, width, height,
        mid_width, mid_height, mid_sinogram_width);

    //Unbind texture reference to free resource
    hipUnbindTexture(texreference);

    // Copy result matrix from device to host
    hipMemcpy( output_host, output_dev, size_result, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_sinogram_float);
    hipFree(output_dev);
    hipFreeArray(carray);
    /* Export image data. */

    for(j = 0; j < width; j++){
        for(i = 0; i < height; i++){
            fprintf(outputFile, "%e ",output_host[j*width + i]);
        }
        fprintf(outputFile, "\n");
    }


    /* Cleanup: Free host memory, close files. */
    free(sinogram_host);
    free(output_host);

    fclose(outputFile);

    return 0;
}


